#include "hip/hip_runtime.h"
#include "pyspace.h"
#include <math.h>

#define NORM2(X, Y, Z) X*X + Y*Y + Z*Z

__device__
void calculate_force(double* x_old, double* y_old, double* z_old, double* m,
        double x_i, double y_i, double z_i,
        double& a_x, double& a_y, double& a_z,
        int num_planets, double eps2, double G)
{
    double r_x_j, r_y_j, r_z_j;
    double x_ji, y_ji, z_ji;
    double m_j;

    double cnst;
    double dist_ij;

    for(int j=0; j<num_planets; j++)
    {
        r_x_j = x_old[j];
        r_y_j = y_old[j];
        r_z_j = z_old[j];

        m_j = m[j];

        x_ji = r_x_j - x_i;
        y_ji = r_y_j - y_i;
        z_ji = r_z_j - z_i;

        dist_ij = sqrt(eps2 + NORM2(x_ji, y_ji, z_ji));

        if(dist_ij == 0)
            return;

        cnst = (G*m_j/(dist_ij*dist_ij*dist_ij));

        a_x += x_ji*cnst;
        a_y += y_ji*cnst;
        a_z += z_ji*cnst;
    }

}


__global__
void brute_force_kernel(double* x, double* y, double* z,
        double* x_old, double* y_old, double* z_old,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z,
        double* m, double G, double dt, int num_planets)
{
    double eps2 = eps*eps;

    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id > num_planets)
        return;
    
    //Update id'th planet

    double a_x_i = a_x[i];
    double a_y_i = a_y[i];
    double a_z_i = a_z[i];

    calculate_force(x_old, y_old, z_old, m,
            x_old[id], y_old[id], z_old[id],
            a_x[id], a_y[id], a_z[id],
            num_planets, eps2, G);

    x[i] += v_x[i]*dt + a_x_i*0.5*dt*dt;
    y[i] += v_y[i]*dt + a_y_i*0.5*dt*dt;
    z[i] += v_z[i]*dt + a_z_i*0.5*dt*dt;

    v_x[i] += (a_x_i + a_x[i])*0.5*dt;
    v_y[i] += (a_y_i + a_y[i])*0.5*dt;
    v_z[i] += (a_z_i + a_z[i])*0.5*dt;
}


__host__
void brute_force_gpu_update(double* x, double* y, double* z,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z,
        double* m, double G, double dt, int num_planets, double eps)
{
    if( hipMalloc((void**)&dev_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_x_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_y_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_z_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_m, num_planets*sizeof(double)) != hipSuccess   )
    {
        fprintf(stderr, "ERROR: hipMalloc failed!");
        exit(0);
    }

    if( hipMemcpy(dev_x, x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_y, y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_z, z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_x_old, x_old, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_y_old, y_old, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_z_old, z_old, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_x, v_x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_y, v_y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_z, v_z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_x, a_x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_y, a_y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_z, a_z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_m, m, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess )
    {
        fprintf(stderr, "ERROR: hipMemcpy from host to device failed!");
        exit(0);
    }

    brute_force_kernel<<<num_planets/1024 + 1, 1024>>>(dev_x, dev_y, dev_z,
            dev_x_old, dev_y_old, dev_z_old,
            dev_v_x, dev_v_y, dev_v_z,
            dev_a_x, dev_a_y, dev_a_z,
            dev_m, G, dt, num_planets);

    if( hipMemcpy(dev_x, x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_y, y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_z, z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_x_old, x_old, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_y_old, y_old, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_z_old, z_old, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_v_x, v_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_v_y, v_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_v_z, v_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_a_x, a_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_a_y, a_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_a_z, a_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(dev_m, m, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess )
    {
        fprintf(stderr, "ERROR: hipMemcpy from device to host failed!");
        exit(0);
    }

}

