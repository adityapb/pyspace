#include "hip/hip_runtime.h"
#include "pyspace.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#define NORM2(X, Y, Z) X*X + Y*Y + Z*Z

__device__
void calculate_force_device(double* x_old, double* y_old, double* z_old, double* m,
        double x_i, double y_i, double z_i, int i,
        double& a_x, double& a_y, double& a_z,
        int num_planets, double eps2, double G)
{
    double r_x_j, r_y_j, r_z_j;
    double x_ji, y_ji, z_ji;
    double m_j;

    double cnst;
    double dist_ij;

    int j;
    for(j=0; j<num_planets; j++)
    {
        if(j == i)
            continue;

        r_x_j = x_old[j];
        r_y_j = y_old[j];
        r_z_j = z_old[j];

        m_j = m[j];

        x_ji = r_x_j - x_i;
        y_ji = r_y_j - y_i;
        z_ji = r_z_j - z_i;

        dist_ij = sqrt(eps2 + NORM2(x_ji, y_ji, z_ji));

        cnst = (G*m_j/(dist_ij*dist_ij*dist_ij));

        a_x += x_ji*cnst;
        a_y += y_ji*cnst;
        a_z += z_ji*cnst;
    }

}


__global__
void brute_force_kernel(double* x, double* y, double* z,
        double* x_old, double* y_old, double* z_old,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z,
        double* m, double G, double dt, int num_planets, double eps)
{
    double eps2 = eps*eps;

    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id >= num_planets)
        return;
    
    //Update id'th planet

    double a_x_i = a_x[id];
    double a_y_i = a_y[id];
    double a_z_i = a_z[id];

    double temp_a_x = 0;
    double temp_a_y = 0;
    double temp_a_z = 0;

    calculate_force_device(x_old, y_old, z_old, m,
            x_old[id], y_old[id], z_old[id], int id,
            temp_a_x, temp_a_y, temp_a_z,
            num_planets, eps2, G);

    a_x[id] = temp_a_x;
    a_y[id] = temp_a_y;
    a_z[id] = temp_a_z;

    x[id] += v_x[id]*dt + a_x_i*0.5*dt*dt;
    y[id] += v_y[id]*dt + a_y_i*0.5*dt*dt;
    z[id] += v_z[id]*dt + a_z_i*0.5*dt*dt;

    v_x[id] += (a_x_i + a_x[id])*0.5*dt;
    v_y[id] += (a_y_i + a_y[id])*0.5*dt;
    v_z[id] += (a_z_i + a_z[id])*0.5*dt;
}


__host__
void malloc_device(double* x, double* y, double* z,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z, double* m,
        double* dev_x, double* dev_y, double* dev_z,
        double* dev_x_old, double* dev_y_old, double* dev_z_old,
        double* dev_v_x, double* dev_v_y, double* dev_v_z,
        double* dev_a_x, double* dev_a_y, double* dev_a_z, 
        double* dev_m, int num_planets)
{
    //cuda Malloc and set dev ptrs
    if( hipMalloc((void**)&dev_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_x_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_y_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_z_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_m, num_planets*sizeof(double)) != hipSuccess   )
    {
        fprintf(stderr, "ERROR: hipMalloc failed!");
        exit(0);
    }

    if( hipMemcpy(dev_x, x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_y, y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_z, z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_x, v_x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_y, v_y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_z, v_z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_x, a_x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_y, a_y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_z, a_z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_m, m, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess )
    {
        fprintf(stderr, "ERROR: hipMemcpy from host to device failed!");
        exit(0);
    }

}

__host__
void memcpy_to_host(double* x, double* y, double* z,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z, double* m,
        double* dev_x, double* dev_y, double* dev_z,
        double* dev_v_x, double* dev_v_y, double* dev_v_z,
        double* dev_a_x, double* dev_a_y, double* dev_a_z,
        double* dev_m, int num_planets)
{
    //Copy data to host

    if( hipMemcpy(x, dev_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(y, dev_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(z, dev_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess || 
        hipMemcpy(v_x, dev_v_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(v_y, dev_v_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(v_z, dev_v_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(a_x, dev_a_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(a_y, dev_a_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(a_z, dev_a_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess )
    {
        fprintf(stderr, "ERROR: hipMemcpy from device to host failed!\n");
        exit(0);
    }

}

__host__
void free_device(double* dev_x, double* dev_y, double* dev_z,
        double* dev_x_old, double* dev_y_old, double* dev_z_old,
        double* dev_v_x, double* dev_v_y, double* dev_v_z,
        double* dev_a_x, double* dev_a_y, double* dev_a_z,
        double* dev_m)
{
    if( hipFree(dev_x) != hipSuccess ||
        hipFree(dev_y) != hipSuccess ||
        hipFree(dev_z) != hipSuccess ||
        hipFree(dev_x_old) != hipSuccess ||
        hipFree(dev_y_old) != hipSuccess ||
        hipFree(dev_z_old) != hipSuccess ||
        hipFree(dev_v_x) != hipSuccess ||
        hipFree(dev_v_y) != hipSuccess ||
        hipFree(dev_v_z) != hipSuccess ||
        hipFree(dev_a_x) != hipSuccess ||
        hipFree(dev_a_y) != hipSuccess ||
        hipFree(dev_a_z) != hipSuccess ||
        hipFree(dev_m) != hipSuccess  )
    {
        fprintf(stderr, "ERROR: hipFree failed!");
        exit(0);
    }
}

__host__
void brute_force_gpu_update(double* dev_x, double* dev_y, double* dev_z,
        double* dev_x_old, double* dev_y_old, double* dev_z_old,
        double* dev_v_x, double* dev_v_y, double* dev_v_z,
        double* dev_a_x, double* dev_a_y, double* dev_a_z,
        double* dev_m, double G, double dt, int num_planets, double eps)
{
    if( hipMemcpy(dev_x_old, dev_x, num_planets*sizeof(double), hipMemcpyDeviceToDevice) != hipSuccess ||
        hipMemcpy(dev_y_old, dev_y, num_planets*sizeof(double), hipMemcpyDeviceToDevice) != hipSuccess ||
        hipMemcpy(dev_z_old, dev_z, num_planets*sizeof(double), hipMemcpyDeviceToDevice) != hipSuccess  )
    {
        fprintf(stderr, "ERROR: hipMemcpy from device to device failed!\n");
        exit(0);
    }
        
    int num_blocks = ceil(num_planets/256);
    brute_force_kernel<<<num_blocks, 256>>>(dev_x, dev_y, dev_z,
            dev_x_old, dev_y_old, dev_z_old,
            dev_v_x, dev_v_y, dev_v_z,
            dev_a_x, dev_a_y, dev_a_z,
            dev_m, G, dt, num_planets, eps);

    hipError_t err = hipGetLastError();

    if(err != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        exit(0);
    }
}

